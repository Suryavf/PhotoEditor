// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <math.h>

typedef hipfftDoubleComplex Complex;

extern "C" void executeFFT(uchar *h_R , uchar *h_G , uchar *h_B ,
                           uchar *h_Mag, int rows, int cols){
	int i,j;

	// RGB to complex
	int id = 0;
    Complex **cImg=new Complex*[rows];
	for (i=0;i<rows;i++){
        cImg[i] = new Complex[rows];
        for (j=0;j<cols;j++){
            cImg[i][j].x = (   (double)R[id]
            	              + (double)G[id]
            	              + (double)B[id]  )/3.0;
            cImg[i][j].y = 0.0;
            ++id;
        }
    }
    
    // Copy to device
    Complex  *d_cImg;
    hipMalloc((void**) &d_cImg, rows*cols*sizeof(Complex));

    for(i=0; i<rows; ++i){
        hipMemcpy2D(d_cImg + i*cols, sizeof(Complex), cImg[i], sizeof(Complex), sizeof(Complex), cols, hipMemcpyHostToDevice);
    }

	// Create plan
    hipfftHandle  planFFT;
    hipfftPlan2d(&planFFT, rows, cols, HIPFFT_Z2Z);
    
    // Execute FFT
    hipfftExecZ2Z(planFFT,d_cImg, d_cImg, HIPFFT_FORWARD);

    // Copy to host
    Complex *fft = (Complex*)malloc(rows*cols*sizeof(Complex));
    hipMemcpy(fft, d_cImg, sizeof(Complex)*rows*cols , hipMemcpyDeviceToHost);

    

    

    // Free memory
        free(   fft);
        free(  cImg);
    hipFree(d_cImg);
    hipfftDestroy(planFFT);

}

